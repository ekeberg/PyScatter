#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>

#define PI 3.141592653589793f

__global__ void cuda_calculate_scattering(
    complex<float> * const diffraction,
    const int size,
    const float * const S,
    const float * const coord,
    const int coord_length,
    const float * const occupancy
)
{
    int S_index = blockIdx.x * blockDim.x + threadIdx.x;

    float dotp;
    complex<float> res;
    float res_real, res_imag;

    if (S_index < size) {
        for (int coord_index = 0; coord_index < coord_length; coord_index++) {
            dotp = coord[coord_index*3 + 0] * S[size*0 + S_index] +
                coord[coord_index*3 + 1] * S[size*1 + S_index] + 
                coord[coord_index*3 + 2] * S[size*2 + S_index];

            sincosf(2.0f * PI * dotp, &res_imag, &res_real);
            res_real *= occupancy[coord_index];
            res_imag *= occupancy[coord_index];
            res = complex<float>(res_real, res_imag);
            //res *= occupancy[coord_index];

            diffraction[S_index] += res;
        }
    }
}

// # for coord, occupancy in zip(element_coords, element_occupancy):
// #     coord_slice = (slice(None), ) + (None, )*len(S.shape[1:])
// #     dotp = (coord[coord_slice] * S).sum(axis=0)
// #     element_diff += (occupancy * numpy.exp(2j * numpy.pi * dotp))
